#include "hip/hip_runtime.h"


#include <stdio.h>
#include "lodepng.h"

__global__ void rectify(unsigned char * d_output_image, unsigned char * d_input_image) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	d_output_image[4 * idx + 0] = d_input_image[4 * idx + 0] < 127 ? 127 : d_input_image[4 * idx + 0];
	d_output_image[4 * idx + 1] = d_input_image[4 * idx + 1] < 127 ? 127 : d_input_image[4 * idx + 1];
	d_output_image[4 * idx + 2] = d_input_image[4 * idx + 2] < 127 ? 127 : d_input_image[4 * idx + 2];
	d_output_image[4 * idx + 3] = 255;
}

int main(int argc, char** argv) {
	// get input args
	char * input_filename = argv[1];
	char * output_filename = argv[2];

	printf("Now rectifying to\n", input_filename);

	// load input image from png
	unsigned error;
	unsigned char * h_input_image, * h_output_image;
	unsigned width, height;

	error = lodepng_decode32_file(&h_input_image, &width, &height, input_filename);
	if(error) printf("error %u: %s\n", error, lodepng_error_text(error));
	
	const int IMAGE_SIZE = width * height;
	const int IMAGE_BYTES = IMAGE_SIZE * 4 * sizeof(unsigned char);

	h_output_image = (unsigned char *) malloc(IMAGE_BYTES);

	// declare GPU memory pointers
	unsigned char * d_input_image;
	unsigned char * d_output_image;

	// allocate GPU memory
	hipMalloc(&d_input_image, IMAGE_BYTES);
	hipMalloc(&d_output_image, IMAGE_BYTES);

	// transfer image to GPU
	hipMemcpy(d_input_image, h_input_image, IMAGE_BYTES, hipMemcpyHostToDevice);

	// launch kernel
	int block_size = 1024;
	rectify<<<block_size, IMAGE_SIZE / block_size>>>(d_output_image, d_input_image);
	int remainder = IMAGE_SIZE % block_size;
	hipMemcpy(h_output_image, d_output_image, IMAGE_BYTES, hipMemcpyDeviceToHost);

	// process the remainder on CPU
	for (int idx = IMAGE_SIZE - remainder; idx < IMAGE_SIZE; idx++) {
		h_output_image[4 * idx + 0] = h_input_image[4 * idx + 0] < 127 ? 127 : h_input_image[4 * idx + 0];
		h_output_image[4 * idx + 1] = h_input_image[4 * idx + 1] < 127 ? 127 : h_input_image[4 * idx + 1];
		h_output_image[4 * idx + 2] = h_input_image[4 * idx + 2] < 127 ? 127 : h_input_image[4 * idx + 2];
		h_output_image[4 * idx + 3] = 255;
	}

	lodepng_encode32_file(output_filename, h_output_image, width, height);

	hipFree(d_input_image);
	hipFree(d_output_image);
}