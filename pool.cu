#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include "lodepng.h"

float timedifference_msec(struct timeval t0, struct timeval t1) {
    return (t1.tv_sec - t0.tv_sec) * 1000.0f + (t1.tv_usec - t0.tv_usec) / 1000.0f;
}

#define MAX(a, b) (a > b ? a : b)
#define MAX4(a, b, c, d) (MAX(MAX(a, b), MAX(c, d)))

__global__ void the_pool(unsigned char * d_out, unsigned char * d_in, unsigned width) {
	int i = 64 * blockIdx.x + 2 * threadIdx.x;
	int j = 64 * blockIdx.y + 2 * threadIdx.y;

	unsigned char val00, val01, val10, val11;

	for (int comp = 0; comp < 4; comp++) {
		val00 = d_in[4 * width * i + 4 * j + comp];
		val01 = d_in[4 * width * i + 4 * (j + 1) + comp];
		val10 = d_in[4 * width * (i + 1) + 4 * j + comp];
		val11 = d_in[4 * width * (i + 1) + 4 * (j + 1) + comp];

		unsigned char max_value = MAX4(val00, val01, val10, val11);
		d_out[1 * width * i + 2 * j + comp] = max_value;
	}

}

void process(char* input_filename, char* output_filename) {
	unsigned error;
	unsigned char *image, *new_image;
	unsigned width, height;

	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if(error) printf("error %u: %s\n", error, lodepng_error_text(error));

	struct timeval stop, start, start_no_copy, stop_no_copy;
	gettimeofday(&start, NULL);

	long int size = width * height * sizeof(unsigned char) * 4;
	new_image = (unsigned char*) malloc(size);

	// printf("Loaded image with width %d and height %d. Random one is %d\n", width, height, image[4* 200 * width + 4 * 100 + 0]);

	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;

	// allocate GPU memory
	hipMalloc(&d_in, size);
	hipMalloc(&d_out, size / 4);

	// transfer the array to the GPU
	hipMemcpy(d_in, image, size, hipMemcpyHostToDevice);
	gettimeofday(&start_no_copy, NULL);

	// launch the kernel
	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid(width / 64, height / 64, 1);
	the_pool<<<dimGrid, dimBlock>>>(d_out, d_in, width);

	gettimeofday(&stop_no_copy, NULL);
	float elapsed_no_copy = timedifference_msec(start_no_copy, stop_no_copy);
	printf("GPU processing took %f ms\n", elapsed_no_copy);
	// copy back the result array to the CPU
	hipMemcpy(new_image, d_out, size / 4, hipMemcpyDeviceToHost);

	// Do the remainder
	int remainder_width = width % 64;
	int remainder_height = height % 64;

	for (int i = height - remainder_height; i < height -1; i+=2) {
		for (int j = 0; j < width - 1; j+=2) {
			unsigned char val00, val01, val10, val11;

			for (int comp = 0; comp < 4; comp++) {
				val00 = image[4 * width * i + 4 * j + comp];
				val01 = image[4 * width * i + 4 * (j + 1) + comp];
				val10 = image[4 * width * (i + 1) + 4 * j + comp];
				val11 = image[4 * width * (i + 1) + 4 * (j + 1) + comp];

				unsigned char max_value = MAX4(val00, val01, val10, val11);
				new_image[1 * width * i + 2 * j + comp] = max_value;
			}
		}
	}

	for (int i = 0; i < height -1; i+=2) {
		for (int j = width - remainder_width; j < width - 1; j+=2) {
			unsigned char val00, val01, val10, val11;

			for (int comp = 0; comp < 4; comp++) {
				val00 = image[4 * width * i + 4 * j + comp];
				val01 = image[4 * width * i + 4 * (j + 1) + comp];
				val10 = image[4 * width * (i + 1) + 4 * j + comp];
				val11 = image[4 * width * (i + 1) + 4 * (j + 1) + comp];

				unsigned char max_value = MAX4(val00, val01, val10, val11);
				new_image[1 * width * i + 2 * j + comp] = max_value;
			}
		}
	}

	gettimeofday(&stop, NULL);
	float elapsed = timedifference_msec(start, stop);
	printf("Pool took %f ms\n", elapsed);

	lodepng_encode32_file(output_filename, new_image, width/2, height/2);

	free(image);
	free(new_image);
}

int main(int argc, char ** argv) {
	char* input_filename = argv[1];
	char* output_filename = argv[2];

	process(input_filename, output_filename);

	return 0;
}
